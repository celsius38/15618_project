#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>


struct GlobalConstants {
    size_t* vertex_degree;
    size_t* vertex_start_index;
    size_t* adj_list;
    size_t minPts;
    size_t N;
};

__constant__ GlobalConstants cuConstParams;

__global__ void
bfs_kernel(size_t* boarder, int* labels, int counter) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j < cuConstParams.N) {
    	if(boarder[j]) {
            boarder[j] = 0;
            labels[j] = counter;
            if(cuConstParams.vertex_degree[j] < cuConstParams.minPts) {
                return;
            }
            size_t start_index = cuConstParams.vertex_start_index[j];
            size_t end_index = start_index + cuConstParams.vertex_degree[j];
            for(size_t neighbor_index = start_index; 
                neighbor_index < end_index; 
                neighbor_index++) {
                size_t neighbor = cuConstParams.adj_list[neighbor_index];
                if(labels[neighbor] <= 0) {
                    boarder[neighbor] = 1;
                }
            }
        }
    }
}

__global__ void
degree_kernel(size_t* vertex_degree, float* points_x, float* points_y, float eps, size_t N) {
    int v = blockIdx.x * blockDim.x + threadIdx.x;
    if (v < N) {
        size_t degree = 0;
        float p1_x = points_x[v];
        float p1_y = points_y[v];
        for(size_t i = 0; i < N; i++){
            float p2_x = points_x[i];
            float p2_y = points_y[i];
            if((p1_x-p2_x)*(p1_x-p2_x) + (p1_y-p2_y)*(p1_y-p2_y) <= eps*eps){
                degree++;
            }
        }
        vertex_degree[v] = degree;
    }
}

void setup(size_t* vertex_degree, size_t* vertex_start_index, size_t* adj_list, size_t minPts, size_t N, size_t adj_list_len) {
    int bytes = sizeof(size_t) * N;
    int adj_list_bytes = sizeof(size_t) * adj_list_len;

    size_t* device_degree;
    size_t* device_start_index;
    size_t* device_adj_list;

    hipMalloc(&device_degree, bytes);
    hipMalloc(&device_start_index, bytes);
    hipMalloc(&device_adj_list, adj_list_bytes);

    hipMemcpy(device_degree, vertex_degree, bytes, hipMemcpyHostToDevice);
    hipMemcpy(device_start_index, vertex_start_index, bytes, hipMemcpyHostToDevice);
    hipMemcpy(device_adj_list, adj_list, adj_list_bytes, hipMemcpyHostToDevice);

    GlobalConstants params;
    params.vertex_degree = device_degree;
    params.vertex_start_index = device_start_index;
    params.adj_list = device_adj_list;
    params.minPts = minPts;
    params.N = N;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstParams), &params, sizeof(GlobalConstants));
}

// TODO: hipFree GlobalConstants

void bfs_cuda(size_t* boarder, int* labels, int counter, size_t N) {
    int bytes = sizeof(size_t) * N;
	int labels_byte = sizeof(int) * N;

    // compute number of blocks and threads per block
    const int threadsPerBlock = 512;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    size_t* device_boarder;
    int* device_labels;

    hipMalloc(&device_boarder, bytes);
    hipMalloc(&device_labels, labels_byte);

    hipMemcpy(device_boarder, boarder, bytes, hipMemcpyHostToDevice);
    hipMemcpy(device_labels, labels, labels_byte, hipMemcpyHostToDevice);

    bfs_kernel<<<blocks, threadsPerBlock>>>(device_boarder, device_labels, counter);

    hipDeviceSynchronize();
    hipMemcpy(boarder, device_boarder, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(labels, device_labels, labels_byte, hipMemcpyDeviceToHost);

    hipFree(device_boarder);
    hipFree(device_labels);
}

void degree_cuda(size_t* vertex_degree, float* points_x, float* points_y, float eps, size_t N) {
    int bytes_degree = sizeof(size_t) * N;
    int bytes_points = sizeof(float) * N;

    // compute number of blocks and threads per block
    const int threadsPerBlock = 512;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    size_t* device_degree;
    float* device_points_x;
    float* device_points_y;

    hipMalloc(&device_degree, bytes_degree);
    hipMalloc(&device_points_x, bytes_points);
    hipMalloc(&device_points_y, bytes_points);

    hipMemcpy(device_points_x, points_x, bytes_points, hipMemcpyHostToDevice);
    hipMemcpy(device_points_y, points_y, bytes_points, hipMemcpyHostToDevice);

    degree_kernel<<<blocks, threadsPerBlock>>>(device_degree, device_points_x, device_points_y, eps, N);

    hipDeviceSynchronize();
    hipMemcpy(vertex_degree, device_degree, bytes_degree, hipMemcpyDeviceToHost);

    hipFree(device_degree);
    hipFree(device_points_x);
    hipFree(device_points_y);

}

